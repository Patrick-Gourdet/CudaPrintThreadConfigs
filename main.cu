#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include "cudal.cuh"
#include <pthread.h>

__global__
void callCuda(cudal *c,int *array)
{
    printf("%d",*(array + 1));

    c->cudalPrint(array);
}
//void threaded(){
//    int array[10] = {0};
//    cudal *c = new cudal();
//    callCuda<<<1,5>>>(c,array);
//    hipDeviceSynchronize();
//}
int main()
{
    int *array;
    auto  SIZE = 80;
    hipMallocManaged((void**)&array,sizeof(int) * 20);
    for(auto i{0}; i < SIZE;i++)
    {
        array[i] = i + 10;
    }
    printf("%d",array[0]);
    dim3 threads(4,2,1);
    dim3 block(2,1,1);
    cudal *c = new cudal();

    callCuda<<<threads,block>>>(c, array);
    hipDeviceSynchronize();

    std::cout << "Hello, World!" << std::endl;
    hipFree(array);
    return 0;
}
