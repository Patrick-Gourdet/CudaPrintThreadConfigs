//
// Created by cplusplus on 8/10/20.
//


#include <stdio.h>

//fonts color
#define FBLACK      "\033[30;"
#define FRED        "\033[31;"
#define FGREEN      "\033[32;"
#define FYELLOW     "\033[33;"
#define FBLUE       "\033[34;"
#define FPURPLE     "\033[35;"
#define D_FGREEN    "\033[6;"
#define FWHITE      "\033[7;"
#define FCYAN       "\x1b[36m"

//background color
#define BBLACK      "40m"
#define BRED        "41m"
#define BGREEN      "42m"
#define BYELLOW     "43m"
#define BBLUE       "44m"
#define BPURPLE     "45m"
#define D_BGREEN    "46m"
#define BWHITE      "47m"

//end color
#define NONE        "\033[0m"
#include <cstdio>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include "cudal.cuh"


__device__
void cudal::cudalPrint(int *array) {
    printf("%d",*(array + 1));

    auto g = blockDim.x * blockIdx.x + threadIdx.x;
    printf("threads in\n threads only:\t x: %d, y: %d, z: %d \n",
            threadIdx.x,threadIdx.y,threadIdx.z);
    if(threadIdx.x > 0 && threadIdx.x < 20)
    printf("ARRAY\n threads only:\t x: %d\n",
           *(array + threadIdx.x));//,array[threadIdx.y],array[threadIdx.z]);
    printf("threads in\n block dimension in x y and z and their multiples:\t block x: %d, block y: %d, block z: %d, x * y dimention: %d, x * y * z block dim %d \n"
            ,blockDim.x , blockDim.y,blockDim.z,blockDim.x * blockIdx.y,blockDim.x * blockIdx.y* blockDim.z );

    printf( "\n\nthreads  in the blocks ?? BLOCK dim BLOCK Idx THREAD idx ?? in xx.x yy.y zz.z\n\n"
           "block Dim x * block Idx x + thread Idx X:\n\t %d\n"
           "block Dim y * block Idx y + thread Idx Y:\n\t %d\n"
           "block Dim z * block Idx z + thread Idx Z:\n\t %d \n",
           blockDim.x * blockIdx.x + threadIdx.x,
           blockDim.y * blockIdx.y + threadIdx.z,
           blockDim.z * blockIdx.z + threadIdx.z);
    printf("\n\nthreads  in the blocks dim idx and thread in xy.x yx.y zy.z\n\n \t|"
           "Dim x Dim y Thread x: %d\n\t|"
           "Dim x Dim z Thread x: %d\n\t|"
           "Dim x Dim y Thread y: %d\n\t|"
           "Dim y Dim z Thread y: %d\n\t|"
           "Dim x Dim y Thread z: %d\n\t|"
           "Dim x Dim y Thread z: %d\n\t|",
           blockDim.x * blockDim.y + threadIdx.x,
           blockDim.x * blockDim.z + threadIdx.x,
           blockDim.x * blockDim.y + threadIdx.y,
           blockDim.y * blockDim.z + threadIdx.y,
           blockDim.x * blockDim.y + threadIdx.z,
           blockDim.y * blockDim.z + threadIdx.z);
    if(blockDim.x * blockDim.y + threadIdx.x > 0 && blockDim.x * blockDim.y + threadIdx.x< 20)
    printf("\n\nARRAY  in the blocks dim idx and thread in xy.x yx.y zy.z\n\n \t|"
           "Dim x Dim y Thread x: %d\n\t|"
           "Dim x Dim z Thread x: %d\n\t|"
           "Dim x Dim y Thread y: %d\n\t|"
           "Dim y Dim z Thread y: %d\n\t|"
           "Dim x Dim y Thread z: %d\n\t|"
           "Dim x Dim y Thread z: %d\n\t|",
    *(array + blockDim.x * blockDim.y + threadIdx.x),
    *(array + blockDim.x * blockDim.z + threadIdx.x),
    *(array + blockDim.x * blockDim.y + threadIdx.y),
    *(array + blockDim.y * blockDim.z + threadIdx.y),
    *(array + blockDim.x * blockDim.y + threadIdx.z),
    *(array + blockDim.y * blockDim.z + threadIdx.z));

}

